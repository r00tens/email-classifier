#include "hip/hip_runtime.h"
#include "CudaTimer.cuh"
#include "NaiveBayesGPU.cuh"

#include "../utils/Timer.hpp"

#include <iostream>
#include <numeric>

template <typename KernelFunc>
void NaiveBayesGPU::calculateBlockAndGridSize(KernelFunc kernel, const size_t dataSize, int& numBlocks, int& blockSize,
                                              size_t dynamicSharedMem, const RoundingStrategy strategy)
{
    constexpr int WARP_SIZE = 32;
    constexpr int ROUND_UP_OFFSET = WARP_SIZE - 1;
    constexpr int ROUND_NEAREST_OFFSET = WARP_SIZE / 2;

    int maxBlockSize = m_gpuInfo.getMaxThreadsPerBlock();
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, dynamicSharedMem, maxBlockSize);

    switch (strategy)
    {
    case ROUND_UP:
        blockSize = ((blockSize + ROUND_UP_OFFSET) / WARP_SIZE) * WARP_SIZE;
        break;
    case ROUND_NEAREST:
        blockSize = ((blockSize + ROUND_NEAREST_OFFSET) / WARP_SIZE) * WARP_SIZE;
        break;
    case ROUND_DOWN:
        blockSize = (blockSize / WARP_SIZE) * WARP_SIZE;
        break;
    }

    numBlocks = GPUInfo::calculateNumBlocks(static_cast<int>(dataSize), blockSize);

    numBlocks = std::max(numBlocks, minGridSize);
}

__global__ void countClassesKernel(int* classCounts, const int* labels, const size_t numSamples)
{
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < numSamples)
    {
        atomicAdd(&classCounts[labels[idx]], 1);
    }
}

__global__ void countFeaturesKernel(int* featureCounts, const int* trainLabels, const size_t* rowPointers,
                                    const size_t* columnIndices, const int* values, const size_t numSamples,
                                    const size_t vocabularySize)
{
    const unsigned long long idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < numSamples)
    {
        const int label = trainLabels[idx];

        const size_t rowStart = rowPointers[idx];
        const size_t rowEnd = rowPointers[idx + 1];

        for (size_t j = rowStart; j < rowEnd; j++)
        {
            const size_t featureIndex = columnIndices[j];
            const int count = values[j];

            atomicAdd(&featureCounts[(label * vocabularySize) + featureIndex], count);
        }
    }
}

__global__ void calculateClassProbabilitiesKernel(double* classProbabilitiesLog, const int* classCounts,
                                                  const size_t totalSamples, const int numClasses)
{
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < numClasses)
    {
        classProbabilitiesLog[idx] = log(static_cast<double>(classCounts[idx]) / static_cast<double>(totalSamples));
    }
}

__global__ void calculateTotalFeatureCountKernel(int* dTotalFeatureCounts, const int* dFeatureCounts,
                                                 const size_t vocabularySize, const int numClasses)
{
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int label = idx / vocabularySize;
    const unsigned int featureIndex = idx % vocabularySize;

    if (label < numClasses && featureIndex < vocabularySize)
    {
        atomicAdd(&dTotalFeatureCounts[label], dFeatureCounts[(label * vocabularySize) + featureIndex]);
    }
}

__global__ void calculateFeatureProbabilitiesKernel(double* dFeatureProbabilitiesLog, const int* dFeatureCounts,
                                                    const int* dTotalFeatureCounts, const size_t vocabularySize,
                                                    const int numClasses)
{
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const size_t label = idx / vocabularySize;
    const size_t featureIndex = idx % vocabularySize;

    if (label < numClasses && featureIndex < vocabularySize)
    {
        const int count = dFeatureCounts[(label * vocabularySize) + featureIndex];
        const int totalFeatureCount = dTotalFeatureCounts[label];
        const double probability = static_cast<double>(count + 1) / static_cast<double>(totalFeatureCount +
            vocabularySize);

        dFeatureProbabilitiesLog[(label * vocabularySize) + featureIndex] = log(probability);
    }
}

// __global__ void predictKernel(double* dLogProbabilities, const size_t* dRowPointers, const size_t* dColumnIndices,
//                               const int* dValues, const double* dClassProbabilitiesLog,
//                               const double* dFeatureProbabilitiesLog, const size_t vocabularySize, const int numClasses,
//                               const int sampleIdx)
// {
//     const unsigned int label = (blockIdx.x * blockDim.x) + threadIdx.x;
//
//     if (label < numClasses)
//     {
//         double logProb = dClassProbabilitiesLog[label];
//
//         const size_t rowStart = dRowPointers[sampleIdx];
//         const size_t rowEnd = dRowPointers[sampleIdx + 1];
//
//         for (size_t i = rowStart; i < rowEnd; ++i)
//         {
//             const size_t featureIndex = dColumnIndices[i];
//             const int count = dValues[i];
//
//             if (featureIndex < vocabularySize)
//             {
//                 logProb += count * dFeatureProbabilitiesLog[(label * vocabularySize) + featureIndex];
//             }
//             else
//             {
//                 logProb += count * log(1.0 / static_cast<double>(vocabularySize + 1));
//             }
//         }
//
//         dLogProbabilities[label] = logProb;
//     }
// }

__global__ void predictKernel(double* dLogProbabilities, const size_t* dRowPointers, const size_t* dColumnIndices,
                              const int* dValues, const double* dClassProbabilitiesLog,
                              const double* dFeatureProbabilitiesLog, const size_t vocabularySize, const int numClasses,
                              const size_t numSamples)
{
    const int sampleIdx = static_cast<int>(blockIdx.y);
    const unsigned int label = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (label < numClasses && sampleIdx < numSamples)
    {
        double logProb = dClassProbabilitiesLog[label];

        const size_t rowStart = dRowPointers[sampleIdx];
        const size_t rowEnd = dRowPointers[sampleIdx + 1];

        for (size_t i = rowStart; i < rowEnd; ++i)
        {
            const size_t featureIndex = dColumnIndices[i];
            const int count = dValues[i];

            if (featureIndex < vocabularySize)
            {
                logProb += count * dFeatureProbabilitiesLog[(label * vocabularySize) + featureIndex];
            }
            else
            {
                logProb += count * log(1.0 / static_cast<double>(vocabularySize + 1));
            }
        }

        dLogProbabilities[(sampleIdx * numClasses) + label] = logProb;
    }
}


void NaiveBayesGPU::train(const std::vector<int>& trainLabels, const std::unordered_map<std::string, int>& vocabulary,
                          const CSRMatrix& featureVectorsCSR)
{
    Timer timer;
    timer.start();

    m_vocabulary = vocabulary;

    const size_t numSamples = trainLabels.size();
    const size_t vocabularySize = vocabulary.size();
    const int numClasses = *std::ranges::max_element(trainLabels) + 1;

    int* dTrainLabels;
    int* dClassCounts;
    size_t* dRowPointers;
    size_t* dColumnIndices;
    int* dValues;
    int* dFeatureCounts;
    double* dClassProbabilitiesLog;
    int* dTotalFeatureCounts;
    double* dFeatureProbabilitiesLog;

    hipMalloc(&dTrainLabels, numSamples * sizeof(int));
    hipMalloc(&dClassCounts, numClasses * sizeof(int));
    hipMalloc(&dRowPointers, featureVectorsCSR.rowPointers.size() * sizeof(size_t));
    hipMalloc(&dColumnIndices, featureVectorsCSR.columnIndices.size() * sizeof(size_t));
    hipMalloc(&dValues, featureVectorsCSR.values.size() * sizeof(int));
    hipMalloc(&dFeatureCounts, numClasses * vocabularySize * sizeof(int));
    hipMalloc(&dClassProbabilitiesLog, numClasses * sizeof(double));
    hipMalloc(&dTotalFeatureCounts, numClasses * sizeof(int));
    hipMalloc(&dFeatureProbabilitiesLog, numClasses * vocabularySize * sizeof(double));

    hipMemcpy(dTrainLabels, trainLabels.data(), numSamples * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dRowPointers, featureVectorsCSR.rowPointers.data(),
               featureVectorsCSR.rowPointers.size() * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(dColumnIndices, featureVectorsCSR.columnIndices.data(),
               featureVectorsCSR.columnIndices.size() * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(dValues, featureVectorsCSR.values.data(), featureVectorsCSR.values.size() * sizeof(int),
               hipMemcpyHostToDevice);

    hipMemset(dClassCounts, 0, numClasses * sizeof(int));
    hipMemset(dFeatureCounts, 0, numClasses * vocabularySize * sizeof(int));
    hipMemset(dTotalFeatureCounts, 0, numClasses * sizeof(int));

    int blockSize;
    int numBlocks;

    CudaTimer cudaTimer;
    cudaTimer.start();

    calculateBlockAndGridSize(countClassesKernel, numSamples, numBlocks, blockSize);
    countClassesKernel<<<numBlocks, blockSize>>>(dClassCounts, dTrainLabels, numSamples);

    hipDeviceSynchronize();

    calculateBlockAndGridSize(countFeaturesKernel, numSamples, numBlocks, blockSize);
    countFeaturesKernel<<<numBlocks, blockSize>>>(dFeatureCounts, dTrainLabels, dRowPointers, dColumnIndices, dValues,
                                                  numSamples, vocabularySize);
    hipDeviceSynchronize();

    calculateBlockAndGridSize(calculateClassProbabilitiesKernel, numClasses, numBlocks, blockSize);
    calculateClassProbabilitiesKernel<<<numBlocks, blockSize>>>(dClassProbabilitiesLog, dClassCounts, numSamples,
                                                                numClasses);
    hipDeviceSynchronize();

    calculateBlockAndGridSize(calculateTotalFeatureCountKernel, numClasses * vocabularySize, numBlocks, blockSize);
    calculateTotalFeatureCountKernel<<<numBlocks, blockSize>>>(dTotalFeatureCounts, dFeatureCounts, vocabularySize,
                                                               numClasses);

    hipDeviceSynchronize();

    calculateBlockAndGridSize(calculateFeatureProbabilitiesKernel, numClasses * vocabularySize, numBlocks, blockSize);
    calculateFeatureProbabilitiesKernel<<<numBlocks, blockSize>>>(dFeatureProbabilitiesLog, dFeatureCounts,
                                                                  dTotalFeatureCounts, vocabularySize, numClasses);

    hipDeviceSynchronize();

    cudaTimer.stop();

    std::vector<int> hClassCountsRaw(numClasses);
    std::vector<double> hClassProbabilitiesLog(numClasses);
    std::vector<int> hFeatureCounts(numClasses * vocabularySize);
    std::vector<int> hTotalFeatureCounts(numClasses);
    std::vector<double> hFeatureProbabilitiesLog(numClasses * vocabularySize);

    hipMemcpy(hClassCountsRaw.data(), dClassCounts, numClasses * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hClassProbabilitiesLog.data(), dClassProbabilitiesLog, numClasses * sizeof(double),
               hipMemcpyDeviceToHost);
    hipMemcpy(hFeatureCounts.data(), dFeatureCounts, numClasses * vocabularySize * sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(hTotalFeatureCounts.data(), dTotalFeatureCounts, numClasses * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hFeatureProbabilitiesLog.data(), dFeatureProbabilitiesLog, numClasses * vocabularySize * sizeof(double),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < numClasses; ++i)
    {
        if (hClassCountsRaw[i] > 0)
        {
            m_classCounts[i] = hClassCountsRaw[i];
        }
    }

    for (int i = 0; i < numClasses; ++i)
    {
        m_classProbabilitiesLog[i] = hClassProbabilitiesLog[i];
    }

    for (int label = 0; label < numClasses; label++)
    {
        for (int featureIndex = 0; featureIndex < vocabularySize; featureIndex++)
        {
            const int count = hFeatureCounts[(label * vocabularySize) + featureIndex];

            if (count > 0)
            {
                m_featureCounts[label][featureIndex] = count;
            }
        }
    }

    for (int label = 0; label < numClasses; ++label)
    {
        for (int featureIndex = 0; featureIndex < vocabularySize; ++featureIndex)
        {
            m_featureProbabilitiesLog[label][featureIndex] = hFeatureProbabilitiesLog[(label * vocabularySize) +
                featureIndex];
        }
    }

    hipFree(dTrainLabels);
    hipFree(dClassCounts);
    hipFree(dRowPointers);
    hipFree(dColumnIndices);
    hipFree(dValues);
    hipFree(dFeatureCounts);
    hipFree(dTotalFeatureCounts);
    hipFree(dClassProbabilitiesLog);

    timer.stop();

    std::cout << "GPU: [DONE] [" << cudaTimer.getTimeInSeconds() << " s] [REAL TIME: " << timer.elapsed_time() <<
        " s]\n";
}

auto NaiveBayesGPU::predictBatch(const std::vector<int>& trainLabels, const CSRMatrix& featureVectorsCSR,
                                 const size_t numSamples) -> std::vector<int>
{
    Timer timer;
    timer.start();

    const int numClasses = *std::ranges::max_element(trainLabels) + 1;
    const size_t vocabularySize = m_vocabulary.size();

    double* dLogProbabilities;
    size_t* dRowPointers;
    size_t* dColumnIndices;
    int* dValues;
    double* dClassProbabilitiesLog;
    double* dFeatureProbabilitiesLog;

    hipMalloc(&dLogProbabilities, numSamples * numClasses * sizeof(double));
    hipMalloc(&dRowPointers, featureVectorsCSR.rowPointers.size() * sizeof(size_t));
    hipMalloc(&dColumnIndices, featureVectorsCSR.columnIndices.size() * sizeof(size_t));
    hipMalloc(&dValues, featureVectorsCSR.values.size() * sizeof(int));
    hipMalloc(&dClassProbabilitiesLog, numClasses * sizeof(double));
    hipMalloc(&dFeatureProbabilitiesLog, numClasses * vocabularySize * sizeof(double));

    hipMemcpy(dRowPointers, featureVectorsCSR.rowPointers.data(),
               featureVectorsCSR.rowPointers.size() * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(dColumnIndices, featureVectorsCSR.columnIndices.data(),
               featureVectorsCSR.columnIndices.size() * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(dValues, featureVectorsCSR.values.data(), featureVectorsCSR.values.size() * sizeof(int),
               hipMemcpyHostToDevice);

    std::vector hClassProbabilitiesLog(numClasses, 0.0);
    std::vector hFeatureProbabilitiesLog(numClasses * vocabularySize, 0.0);

    for (const auto& pair : m_classProbabilitiesLog)
    {
        int classIndex = pair.first;
        double probability = pair.second;

        if (classIndex >= 0 && classIndex < numClasses)
        {
            hClassProbabilitiesLog[classIndex] = probability;
        }
    }

    for (const auto& outerPair : m_featureProbabilitiesLog)
    {
        int classIndex = outerPair.first;
        const auto& innerMap = outerPair.second;

        for (const auto& innerPair : innerMap)
        {
            int wordIndex = innerPair.first;
            double probability = innerPair.second;

            int index = static_cast<int>(classIndex * vocabularySize) + wordIndex;

            hFeatureProbabilitiesLog[index] = probability;
        }
    }

    hipMemcpy(dClassProbabilitiesLog, hClassProbabilitiesLog.data(), numClasses * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(dFeatureProbabilitiesLog, hFeatureProbabilitiesLog.data(), numClasses * vocabularySize * sizeof(double),
               hipMemcpyHostToDevice);

    int blockSize;
    dim3 numBlocks;
    int numBlocksX = static_cast<int>(numBlocks.x);

    calculateBlockAndGridSize(predictKernel, numClasses, numBlocksX, blockSize);

    numBlocks.x = numBlocksX;
    numBlocks.y = numSamples;

    CudaTimer cudaTimer;
    cudaTimer.start();

    predictKernel<<<numBlocks, blockSize>>>(dLogProbabilities, dRowPointers, dColumnIndices, dValues,
                                            dClassProbabilitiesLog, dFeatureProbabilitiesLog, vocabularySize,
                                            numClasses, numSamples);

    hipDeviceSynchronize();

    cudaTimer.stop();

    std::vector<double> hLogProbabilities(numSamples * numClasses);
    hipMemcpy(hLogProbabilities.data(), dLogProbabilities, numSamples * numClasses * sizeof(double),
               hipMemcpyDeviceToHost);

    std::vector<int> predictedClasses(numSamples);
    using diffType = std::vector<double>::difference_type;

    for (int sampleIdx = 0; sampleIdx < numSamples; ++sampleIdx)
    {
        diffType start = static_cast<diffType>(sampleIdx) * numClasses;
        diffType end = static_cast<diffType>(sampleIdx + 1) * numClasses;

        int predictedClass = static_cast<int>(std::distance(hLogProbabilities.begin() + start,
                                                            std::ranges::max_element(
                                                                hLogProbabilities.begin() + start,
                                                                hLogProbabilities.begin() + end)));
        predictedClasses[sampleIdx] = predictedClass;
    }


    hipFree(dLogProbabilities);
    hipFree(dRowPointers);
    hipFree(dColumnIndices);
    hipFree(dValues);
    hipFree(dClassProbabilitiesLog);
    hipFree(dFeatureProbabilitiesLog);

    timer.stop();

    std::cout << "GPU: [DONE] [" << cudaTimer.getTimeInSeconds() << " s] [REAL TIME: " << timer.elapsed_time() <<
        " s]\n";

    return predictedClasses;
}


void NaiveBayesGPU::evaluate(const CSRMatrix& featureVectorsCSR, const std::vector<int>& trueLabels, int positiveClass)
{
    const size_t numSamples = trueLabels.size();

    ClassificationLabels classificationLabels;

    classificationLabels.predictedLabels = predictBatch(trueLabels, featureVectorsCSR, numSamples);
    classificationLabels.trueLabels = trueLabels;

    m_evaluationMetrics.accuracy(classificationLabels);
    m_evaluationMetrics.precision(classificationLabels, positiveClass);
    m_evaluationMetrics.recall(classificationLabels, positiveClass);
    m_evaluationMetrics.f1Score();
}

void NaiveBayesGPU::printEvaluationMetrics() const
{
    m_evaluationMetrics.printEvaluationMetrics();
}

auto NaiveBayesGPU::getVocabulary() const -> std::unordered_map<std::string, int>
{
    return m_vocabulary;
}

auto NaiveBayesGPU::getClassCounts() const -> std::unordered_map<int, int>
{
    return m_classCounts;
}

auto NaiveBayesGPU::getFeatureCounts() const -> std::unordered_map<int, std::unordered_map<int, int>>
{
    return m_featureCounts;
}

auto NaiveBayesGPU::getClassProbabilitiesLog() const -> std::unordered_map<int, double>
{
    return m_classProbabilitiesLog;
}

auto NaiveBayesGPU::getFeatureProbabilitiesLog() const -> std::unordered_map<int, std::unordered_map<int, double>>
{
    return m_featureProbabilitiesLog;
}

auto NaiveBayesGPU::getEvaluationMetrics() const -> EvaluationMetrics
{
    return m_evaluationMetrics;
}
